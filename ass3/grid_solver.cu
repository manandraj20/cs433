#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

#define ROWS_A ((1 << 10) + 2)
#define COLS_A ((1 << 10) + 2)

#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

#define SPAN_PER_THREAD_X 8
#define SPAN_PER_THREAD_Y 8
#define BLOCKS_X ((COLS_A - 2) / (THREADS_PER_BLOCK_X * SPAN_PER_THREAD_X))
#define BLOCKS_Y ((ROWS_A - 2) / (THREADS_PER_BLOCK_Y * SPAN_PER_THREAD_Y))

#define TOL 1e-6
#define ITER_LIMIT 1000

__device__ int count = 0;
__device__ volatile int barrier_flag = 0;

__managed__ float global_diff;

__global__ void init_kernel(float *A)
{
    int col = (threadIdx.x + blockIdx.x * blockDim.x) * SPAN_PER_THREAD_X + 1;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) * SPAN_PER_THREAD_Y + 1;

    for (int i = row; i < row + SPAN_PER_THREAD_Y; i++)
    {
        for (int j = col; j < col + SPAN_PER_THREAD_X; j++)
        {
            A[j + COLS_A * i] = (float)(row * col) / (ROWS_A * COLS_A);
        }
    }
}

__global__ void pad_cols_kernel(float *A)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    A[COLS_A * id] = 0.0;
    A[COLS_A * id + COLS_A - 1] = 0.0;
}

__global__ void pad_rows_kernel(float *A)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    A[id] = 0.0;
    A[COLS_A * (ROWS_A - 1) + id] = 0.0;
}

__global__ void grid_solver_kernel(float *A)
{
    int i, j, done = 0, iters = 0;
    float temp, local_diff;
    int local_sense = 0;
    int last_count;
    int col = (threadIdx.x + blockIdx.x * blockDim.x) * SPAN_PER_THREAD_X + 1;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) * SPAN_PER_THREAD_Y + 1;

    while (!done)
    {
        local_diff = 0.0f;
        if (!blockIdx.x && !blockIdx.y)
        {
            global_diff = 0.0f;
        }
        // cg::grid_group grid = cg::this_grid();
        // grid.sync();
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        if (threadIdx.x == 0)
        {
            last_count = atomicAdd(&count, 1);
            if (last_count == (BLOCKS_X - 1))
            {
                count = 0;
                barrier_flag = local_sense;
            }
        }
        while (barrier_flag != local_sense)
            ;

        for (i = row; i < row + SPAN_PER_THREAD_Y; i++)
        {
            for (j = col; j < col + SPAN_PER_THREAD_X; j++)
            {
                temp = A[j + COLS_A * i];
                A[j + COLS_A * i] = 0.2f * (A[j + COLS_A * i] + A[j + 1 + COLS_A * i] + A[j - 1 + COLS_A * i] + A[j + COLS_A * (i + 1)] + A[j + COLS_A * (i - 1)]);
                local_diff += fabsf(A[j + COLS_A * i] - temp);
            }
        }

        atomicAdd(&global_diff, local_diff);
        // cg::grid_group grid = cg::this_grid();
        // grid.sync();

        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        if (threadIdx.x == 0)
        {
            last_count = atomicAdd(&count, 1);
            if (last_count == (BLOCKS_X - 1))
            {
                count = 0;
                barrier_flag = local_sense;
            }
        }
        while (barrier_flag != local_sense)
            ;

        iters++;
        if (global_diff / (ROWS_A * COLS_A) < TOL || iters >= ITER_LIMIT)
        {
            done = 1;
        }
        // cg::grid_group grid = cg::this_grid();
        // grid.sync();

        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        if (threadIdx.x == 0)
        {
            last_count = atomicAdd(&count, 1);
            if (last_count == (BLOCKS_X - 1))
            {
                count = 0;
                barrier_flag = local_sense;
            }
        }
        while (barrier_flag != local_sense)
            ;
        if(!blockIdx.x && !blockIdx.y)
        {
            printf("Iteration: %d, Error: %f\n", iters, global_diff / (ROWS_A * COLS_A));
        }

    }
}

int main(int argc, char *argv[])
{
    struct timeval tv0, tv1, tv2;
    struct timezone tz0, tz1, tz2;

    float *A;
    hipMallocManaged((void **)&A, ROWS_A * COLS_A * sizeof(float));

    gettimeofday(&tv0, &tz0);

    pad_cols_kernel<<<2, (ROWS_A >> 1)>>>(A);
    pad_rows_kernel<<<2, (COLS_A >> 1)>>>(A);

    dim3 dimBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 dimGrid(BLOCKS_X, BLOCKS_Y);

    init_kernel<<<dimGrid, dimBlock>>>(A);

    hipDeviceSynchronize();
    gettimeofday(&tv1, &tz1);

    grid_solver_kernel<<<dimGrid, dimBlock>>>(A);
    hipDeviceSynchronize();

    gettimeofday(&tv2, &tz2);

    printf("Error: %f\n", global_diff / (ROWS_A * COLS_A));
    printf("Time: %ld microseconds, ", (tv2.tv_sec - tv0.tv_sec) * 1000000 + (tv2.tv_usec - tv0.tv_usec));
    printf("Init time: %ld microseconds, ", (tv1.tv_sec - tv0.tv_sec) * 1000000 + (tv1.tv_usec - tv0.tv_usec));
    printf("Compute time: %ld microseconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000 + (tv2.tv_usec - tv1.tv_usec));

    hipFree(A);
    return 0;
}
