#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__global__ void initialize_1D(float *A, float *x, int n, int span)
{
    int i, j;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    for (j = 0; j < n; j++)
        for (i = span * id; i < span * (id + 1); i++)
        {

            A[j + n*i] = 1;
        }
    for (i = span * id; i < span * (id + 1); i++)
    {
        x[i] = 1;
    }
}

__global__ void initialize_2D(float **A, float *x, int n, int span)
{
    int i, j;
    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;

    for (i = span * id_x; i < span * (id_x + 1); i++)
    {
        x[i] = 1;
    }
    for(j = span * id_y; j < span * (id_y + 1); j++)
        for (i = span * id_x; i < span * (id_x + 1); i++)
        {
            A[j][i] = 1;
        }
}

__global__ void solve(float *A, float *x, float *y,  int n, int span)
{
    // solve for y = Ax
    int i, j;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    for (i = span * id; i < span * (id + 1); i++)
    {
        y[i] = 0;
        for (j = 0; j < n; j++)
        {
            y[i] += A[n*i + j] * x[j];
        }
    }
}

int main(int argc, char *argv[])
{
    float *A, *x, *y;
    int n;
    int nthreads;
    struct timeval tv0, tv2, tv1;
    struct timezone tz0, tz2, tz1;


    if (argc != 3)
    {
        printf("Need number of threads.\n");
        exit(1);
    }
    nthreads = atoi(argv[1]);
    assert((nthreads & (nthreads - 1)) == 0);

    n = atoi(argv[2]);
    assert((n & (n - 1)) == 0);
    int device = -1;
    hipGetDevice(&device);
    hipMallocManaged((void **)&A, n *n* sizeof(float));
    // hipMemAdvise(A, n * sizeof(float *), hipMemAdviseSetAccessedBy, device);
    // for (int i = 0; i < n; i++)
    // {
    //     hipMallocManaged((void **)&A[i], n * sizeof(float));
    //     // hipMemAdvise(A[i], n * sizeof(float),hipMemAdviseSetAccessedBy, device);
    // }

    hipMallocManaged((void **)&x, n * sizeof(float));
    // hipMemAdvise(x, n * sizeof(float), hipMemAdviseSetAccessedBy, device);

    gettimeofday(&tv0, &tz0);

    // dim3 grid_dims, block_dims;
    // grid_dims.x = n>>4;
    // block_dims.x = 16;
    //TODO:  make to sure handle the case when y is limited to 1024
    // grid_dims.y = n>>4;
    // block_dims.y = 16;



    nthreads = min(n, nthreads);
    if (nthreads < 32)
    {
        initialize_1D<<<1, nthreads>>>(A, x, n, n / nthreads);
    }
    else
    {
        initialize_1D<<<nthreads/8, 8>>>(A, x, n, n / nthreads);
        // initialize_2D<<<grid_dims, block_dims>>>(A, x, n, n / nthreads);
    }
    hipDeviceSynchronize();
    
    hipMallocManaged((void**)&y, sizeof(float)*n);
    hipMemAdvise(x, sizeof(float)*n, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(A, n *n* sizeof(float),hipMemAdviseSetReadMostly,0);
    // for (int i = 0; i < n; i++)
    // {
    //     hipMemAdvise(A[i], n * sizeof(float),hipMemAdviseSetReadMostly,0);
    // }
    gettimeofday(&tv1, &tz1);
    
    if(nthreads < 16)
    {
        solve<<<1, nthreads>>>(A, x, y, n, n / nthreads);
    }
    else
    {
        solve<<<nthreads / 8, 8>>>(A, x, y, n, n / nthreads);
    }
    hipDeviceSynchronize();
    gettimeofday(&tv2, &tz2);

    printf("Random element: %lf, time: %ld microseconds\n", y[random() % n], (tv2.tv_sec - tv1.tv_sec) * 1000000 + (tv2.tv_usec - tv1.tv_usec));
    // printf("Random element: %lf, time: %ld microseconds\n", A[random() % n], (tv2.tv_sec - tv0.tv_sec) * 1000000 + (tv2.tv_usec - tv0.tv_usec));

    // free the resources
    hipFree(A);
    hipFree(x);
    hipFree(y);

    return 0;
}
